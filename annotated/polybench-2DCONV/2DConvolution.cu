/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

#define SMALL_FLOAT_VAL 0.00000001f

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0)
    printf("Error return from gettimeofday: %d", stat);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

float absVal(float a) {
  if (a < 0) {
    return (a * -1);
  } else {
    return a;
  }
}

float percentDiff(double val1, double val2) {
  if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01)) {
    return 0.0f;
  }

  else {
    return 100.0f *
           (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
  }
}

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define NI 4096
#define NJ 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void conv2D(DATA_TYPE *A, DATA_TYPE *B) {
  int i, j;
  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +0.2;
  c21 = +0.5;
  c31 = -0.8;
  c12 = -0.3;
  c22 = +0.6;
  c32 = -0.9;
  c13 = +0.4;
  c23 = +0.7;
  c33 = +0.10;

  for (i = 1; i < NI - 1; ++i) // 0
  {
    for (j = 1; j < NJ - 1; ++j) // 1
    {
      B[i * NJ + j] =
          c11 * A[(i - 1) * NJ + (j - 1)] + c12 * A[(i + 0) * NJ + (j - 1)] +
          c13 * A[(i + 1) * NJ + (j - 1)] + c21 * A[(i - 1) * NJ + (j + 0)] +
          c22 * A[(i + 0) * NJ + (j + 0)] + c23 * A[(i + 1) * NJ + (j + 0)] +
          c31 * A[(i - 1) * NJ + (j + 1)] + c32 * A[(i + 0) * NJ + (j + 1)] +
          c33 * A[(i + 1) * NJ + (j + 1)];
    }
  }
}

void init(DATA_TYPE *A) {
  int i, j;

  for (i = 0; i < NI; ++i) {
    for (j = 0; j < NJ; ++j) {
      A[i * NJ + j] = ((float)rand()) / ((float)RAND_MAX);
    }
  }
}

void compareResults(DATA_TYPE *B, DATA_TYPE *B_outputFromGpu) {
  int i, j, fail;
  fail = 0;

  // Compare a and b
  for (i = 1; i < (NI - 1); i++) {
    for (j = 1; j < (NJ - 1); j++) {
      if (percentDiff(B[i * NJ + j], B_outputFromGpu[i * NJ + j]) >
          PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n",
         PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

__global__ void
Convolution2D_kernel(DATA_TYPE __attribute__((annotate("4096,4096"))) * A,
                     DATA_TYPE __attribute__((annotate("4096,4096"))) * B)
    __attribute__((annotate("512,128:8,32"))) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

  c11 = +0.2;
  c21 = +0.5;
  c31 = -0.8;
  c12 = -0.3;
  c22 = +0.6;
  c32 = -0.9;
  c13 = +0.4;
  c23 = +0.7;
  c33 = +0.10;

  if ((i < NI - 1) && (j < NJ - 1) && (i > 0) && (j > 0)) {
    B[i * NJ + j] =
        c11 * A[(i - 1) * NJ + (j - 1)] + c21 * A[(i - 1) * NJ + (j + 0)] +
        c31 * A[(i - 1) * NJ + (j + 1)] + c12 * A[(i + 0) * NJ + (j - 1)] +
        c22 * A[(i + 0) * NJ + (j + 0)] + c32 * A[(i + 0) * NJ + (j + 1)] +
        c13 * A[(i + 1) * NJ + (j - 1)] + c23 * A[(i + 1) * NJ + (j + 0)] +
        c33 * A[(i + 1) * NJ + (j + 1)];
  }
}

void convolution2DCuda(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *B_outputFromGpu) {
  double t_start, t_end;

  DATA_TYPE *A_gpu;
  DATA_TYPE *B_gpu;

  hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
  hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil(((float)NI) / ((float)block.x)),
            (size_t)ceil(((float)NJ) / ((float)block.y)));
  t_start = rtclock();
  Convolution2D_kernel<<<grid, block>>>(A_gpu, B_gpu);
  hipDeviceSynchronize();
  t_end = rtclock();
  fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start); //);

  hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ,
             hipMemcpyDeviceToHost);

  hipFree(A_gpu);
  hipFree(B_gpu);
}

int main(int argc, char *argv[]) {
  double t_start, t_end;

  DATA_TYPE *A;
  DATA_TYPE *B;
  DATA_TYPE *B_outputFromGpu;

  A = (DATA_TYPE *)malloc(NI * NJ * sizeof(DATA_TYPE));
  B = (DATA_TYPE *)malloc(NI * NJ * sizeof(DATA_TYPE));
  B_outputFromGpu = (DATA_TYPE *)malloc(NI * NJ * sizeof(DATA_TYPE));

  // initialize the arrays
  init(A);

  GPU_argv_init();

  convolution2DCuda(A, B, B_outputFromGpu);

  // t_start = rtclock();
  // conv2D(A, B);
  // t_end = rtclock();
  // fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start); //);

  // compareResults(B, B_outputFromGpu);

  free(A);
  free(B);
  free(B_outputFromGpu);

  return 0;
}
