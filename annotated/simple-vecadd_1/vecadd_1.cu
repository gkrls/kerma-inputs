
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecAdd(__attribute__((annotate("100000"))) double *a,
                       __attribute__((annotate("100000"))) double *b,
                       __attribute__((annotate("100000"))) double *c,
                       __attribute__((annotate("100000"))) int n)
    __attribute__((annotate("98:1024"))) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < n)
    c[id] = a[id] + b[id];
}

int main(int argc, char *argv[]) {
  int n = 100000;

  double *h_a, *h_b, *h_c;

  double *d_a;
  double *d_b;
  double *d_c;

  size_t bytes = n * sizeof(double);

  h_a = (double *)malloc(bytes);
  h_b = (double *)malloc(bytes);
  h_c = (double *)malloc(bytes);

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  for (int i = 0; i < n; i++) {
    h_a[i] = sin(i) * sin(i);
    h_b[i] = cos(i) * cos(i);
  }

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  int blockSize, gridSize;

  blockSize = 1024;
  gridSize = (int)ceil((float) n / blockSize);

  vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  double sum = 0;
  for (int i = 0; i < n; i++)
    sum += h_c[i];
  printf("final result: %f\n", sum / n);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}